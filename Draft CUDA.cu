#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <ctime>

#define L 300
#define N (L*L)
#define J 1.00
#define IT 5e8 // Number of iterations
#define NTHREADS 256 // Number of GPU threads (This is a starting point and should be tuned for the specific GPU hardware)

__device__ int get_index(int row, int col) {
    return (row * L + col) % N;
}

__device__ int delta_energy(int* lattice, int r, int c) {
    // Compute the change in energy for flipping the spin at (r, c).
    int sum = lattice[get_index((r-1+L)%L, c)] 
            + lattice[get_index((r+1)%L, c)] 
            + lattice[get_index(r, (c-1+L)%L)] 
            + lattice[get_index(r, (c+1)%L)];
    return 2 * lattice[get_index(r, c)] * sum;
}

__global__ void flip_spins(int* lattice, float* prob, float* energy, int* M, hiprandState* states) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;

    int r = idx / L;
    int c = idx % L;
    int delta = delta_energy(lattice, r, c);
    float rnd = hiprand_uniform(&states[idx]);

    if (delta <= 0 || (delta == 4 && rnd < prob[0]) || (delta == 8 && rnd < prob[1])) {
        lattice[get_index(r, c)] *= -1;
        atomicAdd(energy, 2 * delta * J);
        atomicAdd(M, 2 * lattice[get_index(r, c)]);
    }
}

__global__ void setup_rand_kernel(hiprandState *state, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}

void initialize_lattice(int* lattice, float& energy, int& M, hiprandState* states) {
    // Randomly initialize lattice on GPU using CUDA and cuRAND.
    // Energy and M are updated on the host side based on the initial lattice state.
    // Implement this kernel to update lattice with random values and calculate corresponding energy and M.
}

__global__ void initialize_lattice_kernel(int* lattice, float* energy, int* M, hiprandState* states) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;

    // Generate a random state
    hiprandState localState = states[idx];
    float randVal = hiprand_uniform(&localState);
    states[idx] = localState; // Update the state for the next usage

    // Initialize with a random spin value
    lattice[idx] = randVal >= 0.5f ? 1 : -1;

    // Calculate contribution to energy and magnetization from the current spin
    atomicAdd(M, lattice[idx]);

    __syncthreads();

    if (threadIdx.x == 0) {
        // Only one thread calculates the energy to avoid race conditions
        float local_energy = 0.0f;
        for (int i = 0; i < N; i++) {
            int r = i / L;
            int c = i % L;
            local_energy -= J * lattice[i] * (
                lattice[get_index((r+1)%L, c)] +
                lattice[get_index(r, (c+1)%L)]
            );
        }
        atomicAdd(energy, local_energy);
    }
}
int main() {
    // Allocate memory for lattice on the GPU.
    int* dev_lattice;
    hipMalloc((void**)&dev_lattice, N * sizeof(int));

    // Allocate memory for cuRAND states on the GPU.
    hiprandState* dev_states;
    hipMalloc((void**)&dev_states, N * sizeof(hiprandState));

    // Setup CUDA grid and block dimensions.
    dim3 blocksPerGrid((N + NTHREADS - 1) / NTHREADS, 1, 1);
    dim3 threadsPerBlock(NTHREADS, 1, 1);

    // Initialize cuRAND states.
    setup_rand_kernel<<<blocksPerGrid, threadsPerBlock>>>(dev_states, static_cast<unsigned long>(time(nullptr)));
    hipDeviceSynchronize(); // Synchronize to make sure cuRAND states are initialized.

    // Allocate memory on the GPU for energy and magnetization.
    float* dev_energy;
    int* dev_M;
    hipMalloc((void**)&dev_energy, sizeof(float));
    hipMalloc((void**)&dev_M, sizeof(int));

    // Set initial values on the host.
    float energy = 0.0f;
    int M = 0;

    // Copy initial values of energy and magnetization to the GPU.
    hipMemcpy(dev_energy, &energy, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_M, &M, sizeof(int), hipMemcpyHostToDevice);

    // Initialize the lattice on the GPU.
    initialize_lattice_kernel<<<blocksPerGrid, threadsPerBlock>>>(dev_lattice, dev_energy, dev_M, dev_states);
    hipDeviceSynchronize(); // Synchronize to ensure that initialization is complete.

    // Copy the updated energy and magnetization back to the host.
    hipMemcpy(&energy, dev_energy, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&M, dev_M, sizeof(int), hipMemcpyDeviceToHost);

    // Allocate memory for probabilities on the GPU.
    float* dev_probabilities;
    hipMalloc((void**)&dev_probabilities, 2 * sizeof(float));

    // Main simulation loop over temperature.
    float T = 2.0f; // Example temperature
    float prob[2] = {exp(-4 * J / T), exp(-8 * J / T)};
    hipMemcpy(dev_probabilities, prob, 2 * sizeof(float), hipMemcpyHostToDevice);

    // Simulate using the flip_spins kernel.
    flip_spins<<<blocksPerGrid, threadsPerBlock>>>(dev_lattice, dev_probabilities, dev_energy, dev_M, dev_states);
    hipDeviceSynchronize(); // Synchronize to ensure the simulation is complete.

    // Copy the final results back to the host.
    hipMemcpy(&energy, dev_energy, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&M, dev_M, sizeof(int), hipMemcpyDeviceToHost);

    // Output the results.
    std::cout << "Final Energy: " << energy << std::endl;
    std::cout << "Final Magnetization: " << ((float)M)/N << std::endl;

    // Cleanup resources.
    hipFree(dev_lattice);
    hipFree(dev_probabilities);
    hipFree(dev_energy);
    hipFree(dev_M);
    hipFree(dev_states);

    return 0;
}
